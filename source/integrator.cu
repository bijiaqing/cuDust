#include "hip/hip_runtime.h"
#include "cudust.cuh"

// =========================================================================================================================

__global__
void ssa_substep_1 (swarm *dev_particle, real *dev_timestep)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if(idx >= 0 && idx < NUM_PAR)
    {
        real dt = *dev_timestep;

        real azi_i   = dev_particle[idx].position.x;
        real rad_i   = dev_particle[idx].position.y;
        real col_i   = dev_particle[idx].position.z;
        real l_azi_i = dev_particle[idx].dynamics.x;
        real v_rad_i = dev_particle[idx].dynamics.y;
        real l_col_i = dev_particle[idx].dynamics.z;

        real rad_1 = rad_i + 0.5*v_rad_i*dt;
        real col_1 = col_i + 0.5*l_col_i*dt / rad_i / rad_1;
        real azi_1 = azi_i + 0.5*l_azi_i*dt / rad_i / rad_1 / sin(col_i) / sin(col_1);

        while (azi_1 >= AZI_MAX) azi_1 -= 2.0*M_PI;
        while (azi_1 <  AZI_MIN) azi_1 += 2.0*M_PI;

        if (rad_1 < RAD_MIN)
        {
            rad_1 = RAD_MAX;
            col_1 = 0.5*M_PI;
            dev_particle[idx].dynamics.x = sqrt(G*M*RAD_MAX);
            dev_particle[idx].dynamics.y = 0.0;
            dev_particle[idx].dynamics.z = 0.0;
        }

        dev_particle[idx].position.x = azi_1;
        dev_particle[idx].position.y = rad_1;
        dev_particle[idx].position.z = col_1;
    }
}

// =========================================================================================================================

__global__
void ssa_substep_2 (swarm *dev_particle, real *dev_timestep, real *dev_optdepth)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if(idx >= 0 && idx < NUM_PAR)
    {
        real dt = *dev_timestep;
        
        real azi_1   = dev_particle[idx].position.x;
        real rad_1   = dev_particle[idx].position.y;
        real col_1   = dev_particle[idx].position.z;
        real l_azi_i = dev_particle[idx].dynamics.x;
        real v_rad_i = dev_particle[idx].dynamics.y;
        real l_col_i = dev_particle[idx].dynamics.z;

        real bigR_1 = rad_1*sin(col_1);

        // get the velocities of gas in the hydrostatic equilibrium state
        real eta_1    = (IDX_TEMP + IDX_SIGMAG - 1.0)*H_REF*H_REF*pow(bigR_1 / R_REF, IDX_TEMP + 1.0) + IDX_TEMP*(1.0 - bigR_1 / rad_1);
        real vg_rad_1 = 0.0;
        real lg_col_1 = 0.0;
        real lg_azi_1 = sqrt(G*M*bigR_1)*sqrt(1.0 + eta_1);

        // calculate the stopping time and the dimensionless time  
        real ts_1 = ST_REF / sqrt(G*M / R_REF / R_REF / R_REF);
        ts_1 *= pow(bigR_1 / R_REF, 1.0 - IDX_SIGMAG - 0.5*IDX_TEMP); // radial gas density and sound speed 
        ts_1 /= exp(-cos(col_1)*cos(col_1) / (2.0*H_REF*H_REF*sin(col_1)*sin(col_1)*pow(bigR_1 / R_REF, IDX_TEMP + 1.0))); // vertical gas density
        real tau_1 = dt / ts_1;

        // get the updated positions of the stars, assuming col_star is always half-PI
        real rad_star_1 = D / (1.0 + Q);                // for the secondary; for the primary: rad_star_1 * Q
        real azi_star_1 = 0.5*sqrt(G*M / D / D / D)*dt; // for the secondary; for the primary: azi_star_1 + M_PI
        real dist_pri_1 = sqrt(rad_1*rad_1 + Q*Q*rad_star_1*rad_star_1 + 2.0*Q*rad_1*rad_star_1*sin(col_1)*cos(azi_1 - azi_star_1));
        real dist_sec_1 = sqrt(rad_1*rad_1 +     rad_star_1*rad_star_1 - 2.0*  rad_1*rad_star_1*sin(col_1)*cos(azi_1 - azi_star_1));

        // retrieve the optical depth of the particle
        real optdepth = get_optdepth(dev_optdepth, azi_1, rad_1, col_1);

        // calculate the gravity components in the spherical coordinates
        real forces_pri_rad_1 = -G*M   / (1.0 + Q) / dist_pri_1 / dist_pri_1 / dist_pri_1 * (rad_1 + Q*rad_star_1*sin(col_1)*cos(azi_star_1 - azi_1));
        real forces_pri_col_1 = -G*M   / (1.0 + Q) / dist_pri_1 / dist_pri_1 / dist_pri_1 * (      + Q*rad_star_1*cos(col_1)*cos(azi_star_1 - azi_1));
        real forces_pri_azi_1 = -G*M   / (1.0 + Q) / dist_pri_1 / dist_pri_1 / dist_pri_1 * (      + Q*rad_star_1           *sin(azi_star_1 - azi_1));

        real forces_sec_rad_1 = -G*M*Q / (1.0 + Q) / dist_sec_1 / dist_sec_1 / dist_sec_1 * (rad_1 -   rad_star_1*sin(col_1)*cos(azi_star_1 - azi_1));
        real forces_sec_col_1 = -G*M*Q / (1.0 + Q) / dist_sec_1 / dist_sec_1 / dist_sec_1 * (      -   rad_star_1*cos(col_1)*cos(azi_star_1 - azi_1));
        real forces_sec_azi_1 = -G*M*Q / (1.0 + Q) / dist_sec_1 / dist_sec_1 / dist_sec_1 * (      -   rad_star_1           *sin(azi_star_1 - azi_1));

        // calculate the external forces and torques (using the updated positions but outdated velocities)
        real ext_forces_rad_1 = (1.0 - BETA_REF*exp(-optdepth))*forces_pri_rad_1                  + forces_sec_rad_1;
        real ext_torque_col_1 = (1.0 - BETA_REF*exp(-optdepth))*forces_pri_col_1*rad_1            + forces_sec_col_1*rad_1;
        real ext_torque_azi_1 = (1.0 - BETA_REF*exp(-optdepth))*forces_pri_azi_1*rad_1*sin(col_1) + forces_sec_azi_1*rad_1*sin(col_1);

        // calculate the extra terms in spherical coordinates (using the updated positions but outdated velocities)
        real extra_rad_1 = l_azi_i*l_azi_i / bigR_1 / bigR_1 / rad_1 + l_col_i*l_col_i / rad_1 / rad_1 / rad_1;
        real extra_col_1 = l_azi_i*l_azi_i / bigR_1 / bigR_1 / sin(col_1) * cos(col_1);

        // calculate the updated velocities
        real v_rad_1 = v_rad_i + ((ext_forces_rad_1 + extra_rad_1)*ts_1 + vg_rad_1 - v_rad_i)*(1.0 - exp(-0.5*tau_1));
        real l_col_1 = l_col_i + ((ext_torque_col_1 + extra_col_1)*ts_1 + lg_col_1 - l_col_i)*(1.0 - exp(-0.5*tau_1));
        real l_azi_1 = l_azi_i + ((ext_torque_azi_1              )*ts_1 + lg_azi_1 - l_azi_i)*(1.0 - exp(-0.5*tau_1));

        // calculate the external forces and torques (using the updated positions and velocities)
        real ext_forces_rad_2 = ext_forces_rad_1; // because here the external forces are indipendent from velocity
        real ext_torque_col_2 = ext_torque_col_1; // because here the external forces are indipendent from velocity
        real ext_torque_azi_2 = ext_torque_azi_1; // because here the external forces are indipendent from velocity

        // calculate the extra terms in spherical coordinates (using the updated positions and velocities)
        real extra_rad_2 = l_azi_1*l_azi_1 / bigR_1 / bigR_1 / rad_1 + l_col_1*l_col_1 / rad_1 / rad_1 / rad_1;
        real extra_col_2 = l_azi_1*l_azi_1 / bigR_1 / bigR_1 / sin(col_1) * cos(col_1);

        // calculate the next-step velocities
        real v_rad_j = v_rad_i + ((ext_forces_rad_2 + extra_rad_2)*ts_1 + vg_rad_1 - v_rad_i)*(1.0 - exp(-tau_1));
        real l_col_j = l_col_i + ((ext_torque_col_2 + extra_col_2)*ts_1 + lg_col_1 - l_col_i)*(1.0 - exp(-tau_1));
        real l_azi_j = l_azi_i + ((ext_torque_azi_2              )*ts_1 + lg_azi_1 - l_azi_i)*(1.0 - exp(-tau_1));

        // calculate the next-step positions
        real rad_j = rad_1 + 0.5*v_rad_j*dt;
        real col_j = col_1 + 0.5*l_col_j*dt / rad_1 / rad_j;
        real azi_j = azi_1 + 0.5*l_azi_j*dt / rad_1 / rad_j / sin(col_1) / sin(col_j);

        // co-rotating with the stars
        azi_j -= sqrt(G*M / D / D / D)*dt;

        while (azi_j >= AZI_MAX) azi_j -= 2.0*M_PI;
        while (azi_j <  AZI_MIN) azi_j += 2.0*M_PI;

        if (rad_j < RAD_MIN)
        {
            rad_j   = RAD_MAX;
            col_j   = 0.5*M_PI;
            l_azi_j = sqrt(G*M*RAD_MAX);
            v_rad_j = 0.0;
            l_col_j = 0.0;
        }

        dev_particle[idx].position.x = azi_j;
        dev_particle[idx].position.y = rad_j;
        dev_particle[idx].position.z = col_j;

        dev_particle[idx].dynamics.x = l_azi_j;
        dev_particle[idx].dynamics.y = v_rad_j;
        dev_particle[idx].dynamics.z = l_col_j;
    }
}