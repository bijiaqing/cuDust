#include "hip/hip_runtime.h"
#include <ctime>            // for std::time_t, std::time, std::ctime
#include <chrono>           // for std::chrono::system_clock
#include <iomanip>          // for std::setw, std::setfill
#include <sstream>          // for std::stringstream
#include <iostream>         // for std::cout, std::endl
#include <sys/stat.h>       // for mkdir

#include "cudust.cuh"
#include "hiprand/hiprand_kernel.h"

std::mt19937 rand_generator;

int main (int argc, char **argv)
{
    int resume;
    real output_timer = 0.0;
    
    std::string fname;
    std::ofstream ofile;
    std::uniform_real_distribution <real> random(0.0, 1.0); // distribution in [0, 1)

    swarm *particle, *dev_particle;

    hipHostMalloc((void**)&particle, sizeof(swarm)*NUM_PAR);
    hipMalloc((void**)&dev_particle, sizeof(swarm)*NUM_PAR);
    
    real *dustdens, *dev_dustdens;
    real *optdepth, *dev_optdepth;

    hipHostMalloc((void**)&dustdens, sizeof(real)*NUM_DIM);
    hipMalloc((void**)&dev_dustdens, sizeof(real)*NUM_DIM);
    hipHostMalloc((void**)&optdepth, sizeof(real)*NUM_DIM);
    hipMalloc((void**)&dev_optdepth, sizeof(real)*NUM_DIM);

    tree *dev_treenode;

    hipMalloc((void**)&dev_treenode, sizeof(tree)*NUM_PAR);

    real *timestep, *dev_timestep;

    hipHostMalloc((void**)&timestep, sizeof(real));
    hipMalloc((void**)&dev_timestep, sizeof(real));

    int *collrate, *dev_collrate;

    hipHostMalloc((void**)&collrate, sizeof(int));
    hipMalloc((void**)&dev_collrate, sizeof(int));

    hiprandState *dev_rngstate;

    hipMalloc((void**)&dev_rngstate, sizeof(hiprandState)*NUM_PAR);

    cukd::box_t<float3> *dev_boundbox;

    hipMalloc((void**)&dev_boundbox, sizeof(cukd::box_t<float3>));

    swarm_tmp *dev_tmp_info;

    hipMalloc((void**)&dev_tmp_info, sizeof(swarm_tmp)*NUM_PAR);

    if (argc <= 1) // no flag, start from the initial condition
	{
        resume = 0;

        real *profile_azi,  *dev_prof_azi;
        real *profile_rad,  *dev_prof_rad;
        real *profile_col,  *dev_prof_col;
        real *profile_size, *dev_prof_size;

        hipHostMalloc((void**)&profile_azi,  sizeof(real)*NUM_PAR);
        hipMalloc((void**)&dev_prof_azi,     sizeof(real)*NUM_PAR);
        hipHostMalloc((void**)&profile_rad,  sizeof(real)*NUM_PAR);
        hipMalloc((void**)&dev_prof_rad,     sizeof(real)*NUM_PAR);
        hipHostMalloc((void**)&profile_col,  sizeof(real)*NUM_PAR);
        hipMalloc((void**)&dev_prof_col,     sizeof(real)*NUM_PAR);
        hipHostMalloc((void**)&profile_size, sizeof(real)*NUM_PAR);
        hipMalloc((void**)&dev_prof_size,    sizeof(real)*NUM_PAR);

        rand_generator.seed(0); // or use rand_generator.seed(std::time(NULL));

        // rand_uniform  (profile_azi,  NUM_PAR, AZI_INIT_MIN,   AZI_INIT_MAX);
        // rand_conv_pow (profile_rad,  NUM_PAR, RAD_INIT_MIN,   RAD_INIT_MAX, IDX_SIGG - 1.0, SMOOTH_RAD, RES_RAD);
        // rand_uniform  (profile_col,  NUM_PAR, COL_INIT_MIN,   COL_INIT_MAX);
        // rand_powerlaw (profile_size, NUM_PAR, SIZE_INIT_MIN,  SIZE_INIT_MAX, -1.5);

        rand_uniform (profile_azi,  NUM_PAR, 0.0, 1.0);
        rand_uniform (profile_rad,  NUM_PAR, 0.0, 1.0);
        rand_uniform (profile_col,  NUM_PAR, 0.0, 1.0);
        rand_uniform (profile_size, NUM_PAR, 1.0e-6, 1.0e-6);

        hipMemcpy(dev_prof_azi,  profile_azi,  sizeof(real)*NUM_PAR, hipMemcpyHostToDevice);
        hipMemcpy(dev_prof_rad,  profile_rad,  sizeof(real)*NUM_PAR, hipMemcpyHostToDevice);
        hipMemcpy(dev_prof_col,  profile_col,  sizeof(real)*NUM_PAR, hipMemcpyHostToDevice);
        hipMemcpy(dev_prof_size, profile_size, sizeof(real)*NUM_PAR, hipMemcpyHostToDevice);

        particle_init <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_prof_azi, dev_prof_rad, dev_prof_col, dev_prof_size);

        hipHostFree(profile_azi);  hipFree(dev_prof_azi);
        hipHostFree(profile_rad);  hipFree(dev_prof_rad);
        hipHostFree(profile_col);  hipFree(dev_prof_col);
        hipHostFree(profile_size); hipFree(dev_prof_size);

        // auto start = std::chrono::system_clock::now();
        // hipDeviceSynchronize();
        // auto end = std::chrono::system_clock::now();
        // std::chrono::duration<double> elapsed_seconds = end - start;
        // std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

        // optdepth_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
        // optdepth_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_optdepth, dev_particle);
        // optdepth_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
        // optdepth_rint <<<BLOCKNUM_RAD, THREADS_PER_BLOCK>>> (dev_optdepth);
        // optdepth_mean <<<BLOCKNUM_AZI, THREADS_PER_BLOCK>>> (dev_optdepth);
        
        // dustdens_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_dustdens);
        // dustdens_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_dustdens, dev_particle);
        // dustdens_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_dustdens);
        
        // velocity_init <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_optdepth);

        // mkdir(OUTPUT_PATH.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);

        // open_txt_file(ofile, OUTPUT_PATH + "variables.txt");
        // save_variable(ofile);

        // hipMemcpy(dustdens, dev_dustdens, sizeof(real)*NUM_DIM, hipMemcpyDeviceToHost);
        // fname = OUTPUT_PATH + "dustdens_" + frame_num(resume) + ".bin";
        // open_bin_file(ofile, fname);
        // save_bin_file(ofile, dustdens, NUM_DIM);

        // hipMemcpy(optdepth, dev_optdepth, sizeof(real)*NUM_DIM, hipMemcpyDeviceToHost);
        // fname = OUTPUT_PATH + "optdepth_" + frame_num(resume) + ".bin";
        // open_bin_file(ofile, fname);
        // save_bin_file(ofile, optdepth, NUM_DIM);

        hipMemcpy(particle, dev_particle, sizeof(swarm)*NUM_PAR, hipMemcpyDeviceToHost);
        fname = OUTPUT_PATH + "particle_" + frame_num(resume) + ".par";
        open_bin_file(ofile, fname);
        save_bin_file(ofile, particle, NUM_PAR);
    }
    // else
    // {
    //     std::stringstream convert{argv[1]};     // set up a stringstream variable named convert, initialized with the input from argv[1]
    //     if (!(convert >> resume)) resume = -1;  // do the conversion, if conversion fails, set resume to a default value

    //     std::ifstream ifile;
    //     fname = OUTPUT_PATH + "particle_" + frame_num(resume) + ".bin";
    //     load_bin_file(ifile, fname);
    //     read_bin_file(ifile, particle, NUM_PAR);
    //     hipMemcpy(dev_particle, particle, sizeof(swarm)*NUM_PAR,  hipMemcpyHostToDevice);

    //     optdepth_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
    //     optdepth_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_optdepth, dev_particle);
    //     optdepth_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
    //     optdepth_rint <<<BLOCKNUM_RAD, THREADS_PER_BLOCK>>> (dev_optdepth);
    // }

    for (int i = 1 + resume; i <= OUTPUT_NUM; i++)
    {
        while (output_timer < OUTPUT_INT)
        {
            treenode_init <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_treenode);
            cukd::buildTree <tree, tree_traits> (dev_treenode, NUM_PAR, dev_boundbox);                                                  // 250 ms!!
            collrate_calc <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_tmp_info, dev_treenode, dev_collrate, dev_boundbox); //  80 ms!!
            hipMemcpy(collrate, dev_collrate, sizeof(int), hipMemcpyDeviceToHost);
            *timestep = -std::log(1.0 - random(rand_generator)) / static_cast<real>(*collrate);
            if (*timestep > DT_MAX) *timestep = DT_MAX;
            if (*timestep > OUTPUT_INT - output_timer) *timestep = OUTPUT_INT - output_timer;
            hipMemcpy(dev_timestep, timestep, sizeof(real), hipMemcpyHostToDevice);

            // ssa_substep_1 <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_timestep);
            // optdepth_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
            // optdepth_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_optdepth, dev_particle);
            // optdepth_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
            // optdepth_rint <<<BLOCKNUM_RAD, THREADS_PER_BLOCK>>> (dev_optdepth);
            // ssa_substep_2 <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_optdepth, dev_timestep);

            dustcoag_calc <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_tmp_info, dev_treenode, dev_timestep, dev_boundbox, dev_rngstate);
            dustsize_updt <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_particle, dev_tmp_info);

            // hipMemcpy(timestep, dev_timestep, sizeof(real), hipMemcpyDeviceToHost);
            output_timer += *timestep;

            std::cout << *timestep << ' ' << output_timer << std::endl;
        }
    
        output_timer = 0.0;
    
        // calculate dustdens grids for each output
        // dustdens_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_dustdens);
        // dustdens_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_dustdens, dev_particle);
        // dustdens_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_dustdens);

        // hipMemcpy(dustdens, dev_dustdens, sizeof(real)*NUM_DIM, hipMemcpyDeviceToHost);
        // fname = OUTPUT_PATH + "dustdens_" + frame_num(i) + ".bin";
        // open_bin_file(ofile, fname);
        // save_bin_file(ofile, dustdens, NUM_DIM);

        // calculate optical depth grids for each output
        // optdepth_init <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
        // optdepth_enum <<<BLOCKNUM_PAR, THREADS_PER_BLOCK>>> (dev_optdepth, dev_particle);
        // optdepth_calc <<<BLOCKNUM_DIM, THREADS_PER_BLOCK>>> (dev_optdepth);
        // optdepth_rint <<<BLOCKNUM_RAD, THREADS_PER_BLOCK>>> (dev_optdepth);

        // hipMemcpy(optdepth, dev_optdepth, sizeof(real)*NUM_DIM, hipMemcpyDeviceToHost);
        // fname = OUTPUT_PATH + "optdepth_" + frame_num(i) + ".bin";
        // open_bin_file(ofile, fname);
        // save_bin_file(ofile, optdepth, NUM_DIM);

        if (i % OUTPUT_PAR == 0)
        {
            hipMemcpy(particle, dev_particle, sizeof(swarm)*NUM_PAR, hipMemcpyDeviceToHost);
            fname = OUTPUT_PATH + "particle_" + frame_num(i) + ".par";
            open_bin_file(ofile, fname);
            save_bin_file(ofile, particle, NUM_PAR);
        }

        std::time_t end_time = std::chrono::system_clock::to_time_t(std::chrono::system_clock::now());
        std::cout << std::setw(3) << std::setfill('0') << i << "/" << std::setw(3) << std::setfill('0') << OUTPUT_NUM << " finished on " << std::ctime(&end_time);
    }
 
    return 0;

}
